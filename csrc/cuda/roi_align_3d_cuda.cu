//  https://github.com/TimothyZero/MedVision/blob/main/medvision/csrc/cuda/roi_align_3d_cuda.cu
//
//  Apache License
//  Version 2.0, January 2004
//  http://www.apache.org/licenses/

//  Licensed under the Apache License, Version 2.0 (the "License");
//  you may not use this file except in compliance with the License.
//  You may obtain a copy of the License at

//     http://www.apache.org/licenses/LICENSE-2.0

//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.


#include "roi_align_3d_cuda_kernel.cuh"

using namespace at;


void ROIAlign3DForwardCUDAKernelLauncher(
    const at::Tensor features, const at::Tensor rois, const float spatial_scale,
    const int sampling_ratio, const int order,
    const int channels,
    const int depth, const int height, const int width,
    const int num_rois,
    const int pooled_depth, const int pooled_height, const int pooled_width,
    at::Tensor output) {
    const int output_size = num_rois * pooled_depth * pooled_height * pooled_width * channels;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        features.scalar_type(), "ROIAlign3DLauncherForward", ([&] {
        const scalar_t *bottom_data = features.contiguous().data<scalar_t>();
        const scalar_t *rois_data = rois.contiguous().data<scalar_t>();
        scalar_t *top_data = output.contiguous().data<scalar_t>();

        roi_align_3d_forward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, bottom_data, rois_data, scalar_t(spatial_scale),
                sampling_ratio, order, channels,
                depth, height, width,
                pooled_depth, pooled_height, pooled_width,
                top_data);
        }));

    AT_CUDA_CHECK(hipGetLastError());
}

void ROIAlign3DBackwardCUDAKernelLauncher(
    const at::Tensor top_grad, const at::Tensor rois, const float spatial_scale,
    const int sampling_ratio, const int order,
    const int channels,
    const int depth, const int height, const int width,
    const int num_rois,
    const int pooled_depth, const int pooled_height, const int pooled_width,
    at::Tensor bottom_grad) {
    const int output_size = num_rois * pooled_depth * pooled_height * pooled_width * channels;
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        top_grad.scalar_type(), "ROIAlign3DLauncherBackward", ([&] {
        const scalar_t *top_diff = top_grad.data<scalar_t>();
        const scalar_t *rois_data = rois.contiguous().data<scalar_t>();
        scalar_t *bottom_diff = bottom_grad.data<scalar_t>();
        roi_align_3d_backward_cuda_kernel<scalar_t>
            <<<GET_BLOCKS(output_size), THREADS_PER_BLOCK>>>(
                output_size, top_diff, rois_data, spatial_scale, sampling_ratio,
                order, channels,
                depth, height, width,
                pooled_depth, pooled_height, pooled_width,
                bottom_diff);
        }));
    AT_CUDA_CHECK(hipGetLastError());
    }