#include "hip/hip_runtime.h"
//  https://github.com/TimothyZero/MedVision/blob/main/medvision/csrc/cuda/nms_3d_cuda.cu
// 
//  Apache License
//  Version 2.0, January 2004
//  http://www.apache.org/licenses/

//  Licensed under the Apache License, Version 2.0 (the "License");
//  you may not use this file except in compliance with the License.
//  You may obtain a copy of the License at

//     http://www.apache.org/licenses/LICENSE-2.0

//  Unless required by applicable law or agreed to in writing, software
//  distributed under the License is distributed on an "AS IS" BASIS,
//  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
//  See the License for the specific language governing permissions and
//  limitations under the License.

//  modified from torchvion 0.3.0


#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>

#include "common.h"

#include <iostream>
#include <vector>


int const threadsPerBlock = sizeof(unsigned long long) * 8;

template <typename T>
__device__ inline float devIoU_3d(T const* const a, T const* const b) {
    T left = max(a[0], b[0]), right = min(a[3], b[3]);
    T top = max(a[1], b[1]), bottom = min(a[4], b[4]);
    T front = max(a[2], b[2]), back = min(a[5], b[5]);
    T width = max(right - left, (T)0), height = max(bottom - top, (T)0);
    T depth = max(back - front, (T)0);
    T interS = width * height * depth;
    T Sa = (a[3] - a[0]) * (a[4] - a[1]) * (a[5] - a[2]);
    T Sb = (b[3] - b[0]) * (b[4] - b[1]) * (b[5] - b[2]);
    return interS / (Sa + Sb - interS);
}

template <typename T>
__global__ void nms_kernel_3d(
    const int n_boxes,
    const float nms_overlap_thresh,
    const T* dev_boxes,
    unsigned long long* dev_mask) {
    const int row_start = blockIdx.y;
    const int col_start = blockIdx.x;

  // if (row_start > col_start) return;

    const int row_size =
        min(n_boxes - row_start * threadsPerBlock, threadsPerBlock);
    const int col_size =
        min(n_boxes - col_start * threadsPerBlock, threadsPerBlock);

    __shared__ T block_boxes[threadsPerBlock * 7];
    if (threadIdx.x < col_size) {
        block_boxes[threadIdx.x * 7 + 0] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 0];
        block_boxes[threadIdx.x * 7 + 1] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 1];
        block_boxes[threadIdx.x * 7 + 2] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 2];
        block_boxes[threadIdx.x * 7 + 3] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 3];
        block_boxes[threadIdx.x * 7 + 4] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 4];
        block_boxes[threadIdx.x * 7 + 5] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 5];
        block_boxes[threadIdx.x * 7 + 6] =
            dev_boxes[(threadsPerBlock * col_start + threadIdx.x) * 7 + 6];
    }

    __syncthreads();

    if (threadIdx.x < row_size) {
        const int cur_box_idx = threadsPerBlock * row_start + threadIdx.x;
        const T* cur_box = dev_boxes + cur_box_idx * 7;
        int i = 0;
        unsigned long long t = 0;
        int start = 0;
        if (row_start == col_start) {
            start = threadIdx.x + 1;
        }
        for (i = start; i < col_size; i++) {
            if (devIoU_3d<T>(cur_box, block_boxes + i * 7) > nms_overlap_thresh) {
                t |= 1ULL << i;
            }
        }
        const int col_blocks = at::cuda::ATenCeilDiv(n_boxes, threadsPerBlock);
        dev_mask[cur_box_idx * col_blocks + col_start] = t;
    }
}

// boxes is a N x 7 tensor
at::Tensor nms_3d_cuda(const at::Tensor boxes, float nms_overlap_thresh) {
    using scalar_t = float;
    AT_ASSERTM(boxes.is_cuda(), "boxes must be a CUDA tensor");
    at::cuda::CUDAGuard device_guard(boxes.device());

    auto scores = boxes.select(1, 6);
    auto order_t = std::get<1>(scores.sort(0, /* descending=*/true));
    auto boxes_sorted = boxes.index_select(0, order_t);

    int boxes_num = boxes.size(0);

    const int col_blocks = at::cuda::ATenCeilDiv(boxes_num, threadsPerBlock);

    at::Tensor mask =
        at::empty({boxes_num * col_blocks}, boxes.options().dtype(at::kLong));
 
    dim3 blocks(col_blocks, col_blocks);
    dim3 threads(threadsPerBlock);
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        boxes_sorted.scalar_type(), "nms_kernel_3d_cuda", [&] {
        nms_kernel_3d<scalar_t><<<blocks, threads, 0, stream>>>(
            boxes_num,
            nms_overlap_thresh,
            boxes_sorted.data<scalar_t>(),
            (unsigned long long*)mask.data<int64_t>());
        });

    AT_CUDA_CHECK(hipStreamSynchronize(stream));

    at::Tensor mask_cpu = mask.to(at::kCPU);
    unsigned long long* mask_host = (unsigned long long*)mask_cpu.data<int64_t>();

    std::vector<unsigned long long> remv(col_blocks);
    memset(&remv[0], 0, sizeof(unsigned long long) * col_blocks);

    at::Tensor keep =
        at::empty({boxes_num}, boxes.options().dtype(at::kLong).device(at::kCPU));
    int64_t* keep_out = keep.data<int64_t>();

    int num_to_keep = 0;
    for (int i = 0; i < boxes_num; i++) {
        int nblock = i / threadsPerBlock;
        int inblock = i % threadsPerBlock;

        if (!(remv[nblock] & (1ULL << inblock))) {
            keep_out[num_to_keep++] = i;
            unsigned long long* p = mask_host + i * col_blocks;
            for (int j = nblock; j < col_blocks; j++) {
                remv[j] |= p[j];
            }
        }
    }

    AT_CUDA_CHECK(hipGetLastError());
    return order_t.index(
        {keep.narrow(/*dim=*/0, /*start=*/0, /*length=*/num_to_keep)
            .to(order_t.device(), keep.scalar_type())});
}